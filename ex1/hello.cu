#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello(char* d_buf)
{
    char str[] = "Hello";
    for(int i = 0; i < 6; i++) {
        d_buf[i] = str[i];
    }
}

int main(int argc, char* argv[])
{
    char buf[100];
    char *d_buf;

    hipMalloc((void **) &d_buf, 100);

    hello<<<1, 1>>>(d_buf);

    hipMemcpy(buf, d_buf, 100, hipMemcpyDeviceToHost);
    hipFree(d_buf);

    printf("%s\n", buf);

    hipDeviceReset();

    return 0;
}
